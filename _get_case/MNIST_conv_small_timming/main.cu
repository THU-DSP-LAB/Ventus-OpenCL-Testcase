// main.cu — 1024x1024x1024 GEMM (C = A * B), tiled shared memory
// build: nvcc -O3 -std=c++17 -arch=sm_89 -lineinfo -Xptxas -O3 -o gemm main.cu
// run:   ./gemm
// ncu:   ncu --kernel-name-base demangled --kernel-name ::gemm_tiled \
//           --set full --metrics sm__inst_executed.avg.per_cycle_active,sm__inst_executed.sum,sm__cycles_active.sum ./gemm

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cmath>

#ifndef N
#define N 1024
#endif

#ifndef TILE
#define TILE 32
#endif

#define CUDA_CHECK(call) do { \
  hipError_t _e = (call); \
  if (_e != hipSuccess) { \
    fprintf(stderr, "CUDA error %d (%s) at %s:%d\n", _e, hipGetErrorString(_e), __FILE__, __LINE__); \
    std::exit(1); \
  } \
} while(0)

__global__ void gemm_tiled(const float* __restrict__ A,
                           const float* __restrict__ B,
                           float* __restrict__ C,
                           int n)
{
    __shared__ float As[TILE][TILE];
    __shared__ float Bs[TILE][TILE];

    const int row = blockIdx.y * TILE + threadIdx.y;
    const int col = blockIdx.x * TILE + threadIdx.x;

    float acc = 0.0f;
    #pragma unroll
    for (int t = 0; t < n; t += TILE) {
        // 每个线程各搬一个元素到共享内存（对齐、合并良好）
        As[threadIdx.y][threadIdx.x] = A[row * n + (t + threadIdx.x)];
        Bs[threadIdx.y][threadIdx.x] = B[(t + threadIdx.y) * n + col];
        __syncthreads();

        #pragma unroll
        for (int k = 0; k < TILE; ++k)
            acc = fmaf(As[threadIdx.y][k], Bs[k][threadIdx.x], acc);

        __syncthreads();
    }
    C[row * n + col] = acc;
}

int main(){
    const int n = N;
    const size_t bytes = size_t(n) * n * sizeof(float);

    float *hA = (float*)malloc(bytes);
    float *hB = (float*)malloc(bytes);
    float *hC = (float*)malloc(bytes);

    // 简单初始化（避免 Host 计算校验的大开销）
    for (int i = 0; i < n*n; ++i) {
        hA[i] = (i % 3) * 0.5f;
        hB[i] = (i % 5) * 0.25f;
    }

    float *dA, *dB, *dC;
    CUDA_CHECK(hipMalloc(&dA, bytes));
    CUDA_CHECK(hipMalloc(&dB, bytes));
    CUDA_CHECK(hipMalloc(&dC, bytes));
    CUDA_CHECK(hipMemcpy(dA, hA, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dB, hB, bytes, hipMemcpyHostToDevice));

    dim3 block(TILE, TILE);
    dim3 grid(n / TILE, n / TILE);

    // 预热
    for (int i = 0; i < 3; ++i) {
        gemm_tiled<<<grid, block>>>(dA, dB, dC, n);
    }
    CUDA_CHECK(hipDeviceSynchronize());

    // 计时
    hipEvent_t t0, t1;
    CUDA_CHECK(hipEventCreate(&t0));
    CUDA_CHECK(hipEventCreate(&t1));
    CUDA_CHECK(hipEventRecord(t0));
    gemm_tiled<<<grid, block>>>(dA, dB, dC, n);
    CUDA_CHECK(hipEventRecord(t1));
    CUDA_CHECK(hipEventSynchronize(t1));
    float ms = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&ms, t0, t1));

    CUDA_CHECK(hipMemcpy(hC, dC, bytes, hipMemcpyDeviceToHost));

    // 理论 FLOPs: 2 * N^3（乘加算两次）
    double gflops = (2.0 * n * n * n) / (ms * 1e6);
    printf("GEMM %dx%dx%d | time = %.3f ms | %.2f GFLOP/s\n", n, n, n, ms, gflops);
    printf("launch: grid=(%d,%d) block=(%d,%d)\n", grid.x, grid.y, block.x, block.y);

    hipEventDestroy(t0); hipEventDestroy(t1);
    hipFree(dA); hipFree(dB); hipFree(dC);
    free(hA); free(hB); free(hC);
    return 0;
}
